#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <chrono>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda/common.hpp>

/* Compile: nvcc -std=c++11 Cuda_implementation.cu -o Cuda -I/usr/local/include/opencv4 -lopencv_core  -lopencv_highgui -lopencv_imgcodecs -lopencv_imgproc  $(pkg-config opencv4 --libs)
   Run: ./Cuda
   Profiler commands:
   nv-nsight-cu-cli ./Cuda
   nsys profile --stats=true --force-overwrite true --show-output true ./Cuda */

#define BLOCK_SIZE 16

using namespace std;
using namespace cv;

/********** Calculating Summation in Point Spread Function (PSF) ***********************/
__global__ void calcPSF(cuda::PtrStepSz<float> outputImg, Size filterSize, int radius,
int * summa, cuda::PtrStepSz<float> summa_mat) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int size = filterSize.height;
    int midx = size/2;
    int midy = size/2;

    if(row<size && col<size){
        outputImg(row, col) = 0.0;
    }
   __syncthreads();

    if(row <= radius+radius && row >= 0 && col <= radius+radius && col >= 0) {
        if((row-radius) * (row-radius) + (col-radius) * (col-radius) <= radius*radius) {
            outputImg(midx+row-radius, midy+col-radius) = 255.0;
            atomicAdd(&summa[0], 255.0);
        }
    }
    __syncthreads();

    if(row <= radius+radius && row >= 0 && col <= radius+radius && col >= 0) {
        if((row-radius) * (row-radius) + (col-radius) * (col-radius) <= radius*radius) {
            summa_mat(0,0) = summa[0];
        }
    }
    __syncthreads();
}

/********** Normalizing Point Spread Function (PSF) **************************/
__global__ void psf_normalize(int summation, cuda::PtrStepSz<float> outputImg){

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int size = outputImg.rows;
    if(summation!=0 && row<size && col<size && outputImg(row,col)!=0.0f){
        outputImg(row,col) = outputImg(row,col)/summation;
    }
    __syncthreads();
}

/********** FFT Shift of Point Spread Function **********************************/
__global__ void fft_shift(cuda::PtrStepSz<float> input_PSF,
cuda::PtrStepSz<float> output_PSF, int N) {

    int sLine = N;
    int sSlice = N * N;

    // Transformations Equations
    int sEq1 = (sSlice + sLine) / 2;
    int sEq2 = (sSlice - sLine) / 2;

    __syncthreads();

    // Thread Index (1D)
    int xThreadIdx = threadIdx.x;
    int yThreadIdx = threadIdx.y;

    __syncthreads();

    // Block Width & Height
    int blockWidth = blockDim.x;
    int blockHeight = blockDim.y;

    __syncthreads();

    // Thread Index (2D)
    int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    int yIndex = blockIdx.y * blockHeight + yThreadIdx;

    __syncthreads();

    // Thread Index Converted into 1D Index
    int index = (yIndex * N) + xIndex;
    __syncthreads();

    if (xIndex < N / 2){
        if (yIndex < N / 2) {
             output_PSF(0,index) = input_PSF(0,index + sEq1);
             __syncthreads();
        }
        else {
            output_PSF(0,index) = input_PSF(0,index - sEq2);
            __syncthreads();
        }
    }
    else {
        if (yIndex < N / 2) {
            output_PSF(0,index) = input_PSF(0,index + sEq2);
            __syncthreads();
        }
        else{
            output_PSF(0,index) = input_PSF(0,index - sEq1);
            __syncthreads();
        }
    }

}

/********** Merge Image/Filter with zero values plane - Complex data type **********/
template< typename T_in,typename T_out>
__global__ void mergefilter(cuda::PtrStepSz<T_in> input,
cuda::PtrStepSz<T_out> output) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < (input.rows*input.cols) && j < (input.rows*input.cols)){
        output(i,j).x =  input(i,j);
        output(i,j).y =  0.0f;
        __syncthreads();

    }
}

/********** Split Image/Filter containing two planes to a single plane ************/
__global__ void splitfilter(cuda::PtrStepSz<float> input,
cuda::PtrStepSz<float> output) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < (input.rows*input.cols) && j < (input.rows*input.cols)){
        output(i,j) = input(i,j+j);
    }
}

/********** Adding and Division as part of restoration process ****************/
__global__ void pow_add_div_filter(cuda::PtrStepSz<float> input,
cuda::PtrStepSz<float> output, double nsr) {

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < (input.rows*input.cols) && j < (input.rows*input.cols)){
        output(i,j) = nsr + (abs(input(i,j))*abs(input(i,j)));
        output(i,j) = input(i,j)/output(i,j);
    }
}

/********** Dot product of image and wiener filter ****************************/
__global__ void mulSpectrums(cuda::PtrStepSz<float2> complexI,
cuda::PtrStepSz<float2> complexH, cuda::PtrStepSz<float2> complexIH) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < (complexI.rows*complexI.cols) && j < (complexI.rows*complexI.cols)){
        float I_real = complexI(i,j).x;
        float I_img = complexI(i,j).y;
        float H_real = complexH(i,j).x;
        float mul_real = I_real * H_real;
        float mul_imag = I_img * H_real;
        complexIH(i,j).x =  mul_real;
        complexIH(i,j).y =  mul_imag;
    }
}

/********** Normalize final restored image - Fits into (0-255) range based on min and max ********************/
__global__ void normalize_img(cuda::PtrStepSz<uint8_t> input,
cuda::PtrStepSz<uint8_t> output, uint8_t min, uint8_t max) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < (input.rows*input.cols) && j < (input.rows*input.cols)){
        uint8_t temp = round(((uint8_t)input(i, j) - min) * (255.0/(max-min)));
        output(i, j) = temp;
    }
    __syncthreads();
}

/********** Calculate correctness of algorithm using MSE and PSNR ****************************/
/* PSNR - Peak Signal to noise Ratio
   MSE - Mean Squared Error */
double getPSNR(const Mat& I1, const Mat& I2, int R, int snr, double *mse, double *psnr)
{
    Mat s1;
    absdiff(I1, I2, s1); // |I1 - I2|
    s1.convertTo(s1, CV_32F);
    s1 = s1.mul(s1);   // |I1 - I2|^2
    Scalar s = sum(s1);
    double sse = s.val[0] + s.val[1] + s.val[2];

    if( sse <= 1e-10)
        return 0;
    else
    {
        *mse =sse /(double)(I1.channels() * I1.total());
        *psnr = 10.0*log10((255*255)/(*mse));
    }
    return 0;
}


int main()
{
    std::chrono::time_point<std::chrono::system_clock> startPSF, endPSF, startNormalize, endNormalize, startfft, endfft, startmerge, endmerge, startimgnorm, endimgnorm, startsplit1, endsplit1, startsplit2, endsplit2, startpadf, endpadf, startmerge2, endmerge2, startmerge3, endmerge3, startmulspec, endmulspec, startdft, enddft, startidft, endidft;

    int R = 2; // Radius of PSF function
    int snr = 105;
    double psnr = 0.0f;
    double mse  = 0.0f;

    Mat imgIn;
    imgIn = imread("final_images/bear_256.png", IMREAD_GRAYSCALE);
    int width = imgIn.size().width;
    int height = imgIn.size().height;

    // Making sure the images are of square size and even sized
    if(width > height)
        if(height % 2 == 0)
            imgIn = imgIn(Range(0,height), Range(0,height));
        else
            imgIn = imgIn(Range(0,height-1), Range(0,height-1));
    else if(height > width)
        if(width % 2 == 0)
            imgIn = imgIn(Range(0,width), Range(0,width));
        else
            imgIn = imgIn(Range(0,width-1), Range(0,width-1));
    Mat blurIn = imgIn.clone();

    /**************** Start - Blur image *******************************************/
    GaussianBlur(imgIn, blurIn, Size(5,5), 0);
    imwrite("final_images/blur_cuda_256.jpg", blurIn);
    /**************** End - Blur image **********************************************/


    /**************** Start - Calculate PSF ********************************************/
    int *d_summa;
    int *summa;
    int summa_size = 1 * 1 * sizeof( int);
    hipMalloc((void **)&d_summa, summa_size);
    summa = ( int *)malloc(summa_size);
    summa[0] = 0;

    Mat imgOut;
    Rect roi = Rect(0, 0, blurIn.cols & -2, blurIn.rows & -2);

    Mat Hw, h(roi.height, roi.width, CV_32FC1),h_host(roi.height, roi.width, CV_32FC1);

    cuda::GpuMat h_cuda, h_shifted(roi.height, roi.width, CV_32FC1);
    h_cuda.upload(h);

    hipMemcpy(d_summa, summa, summa_size, hipMemcpyHostToDevice);

    dim3 dimgrid3(roi.height/BLOCK_SIZE,roi.width/BLOCK_SIZE);
    dim3 dimblock3(BLOCK_SIZE,BLOCK_SIZE);

    cuda::GpuMat summa_cuda(1, 1, CV_32FC1);
    Mat summa_host(1, 1, CV_32FC1);
//
    startPSF = std::chrono::system_clock::now();
    calcPSF<<<dimgrid3,dimblock3>>>(h_cuda, roi.size(), R, d_summa, summa_cuda);
    hipDeviceSynchronize();
    endPSF = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsedtime = endPSF - startPSF;
//
    h_cuda.download(h);
    summa_cuda.download(summa_host);
    int summation = summa_host.at<float>(0,0);
//
    startNormalize = std::chrono::system_clock::now();
    psf_normalize<<<dimgrid3,dimblock3>>>(summation, h_cuda);
    hipDeviceSynchronize();
    endNormalize = std::chrono::system_clock::now();
    elapsedtime += endNormalize - startNormalize;
//
    h_cuda.download(h);
    /**************** End - Calculate PSF ********************************************/


    /**************** Start - Wiener Filter ******************************************/
    int cx = h.cols / 2;
    int cy = h.rows / 2;
    cuda::GpuMat fftshift_input_cuda;
    cuda::GpuMat fftshift_output_cuda(roi.height*roi.width,1,CV_32FC1);

    cuda::GpuMat merge_output_cuda(roi.height, roi.width, CV_32FC2);
    cuda::GpuMat dft_output_cuda(roi.height, roi.width, CV_32FC2);
    cuda::GpuMat split_output_cuda(roi.height, roi.width, CV_32FC1);
    cuda::GpuMat pow_add_div_output_cuda(roi.height, roi.width, CV_32FC1);

    int threads = cx * cy;
    Mat h_flat = h.reshape(1,roi.height*roi.width);
    cuda::GpuMat h_flat_cuda(roi.height*roi.width, 1, CV_32FC1);
    h_flat_cuda.upload(h_flat);
//
    startfft = std::chrono::system_clock::now();
    fft_shift<<<dimgrid3,dimblock3>>>(h_flat_cuda, fftshift_output_cuda, h.cols);
    hipDeviceSynchronize();
    endfft = std::chrono::system_clock::now();
    elapsedtime += endfft - startfft;
//
    Mat fftshift_output_host(roi.height*roi.width,1,CV_32FC1);
    fftshift_output_cuda.download(fftshift_output_host);
    fftshift_output_host = fftshift_output_host.reshape (1, roi.width);
    fftshift_output_cuda = fftshift_output_cuda.reshape (1, roi.width);
//
    startmerge = std::chrono::system_clock::now();
    mergefilter<float, float2><<<dimgrid3,dimblock3>>>(fftshift_output_cuda, merge_output_cuda);
    hipDeviceSynchronize();
    endmerge = std::chrono::system_clock::now();
    elapsedtime += endmerge - startmerge;
//
    Mat merge_output_host(h.rows,h.cols,CV_32FC2);
    merge_output_cuda.download(merge_output_host);

    Mat dft_output_host(h.rows,h.cols,CV_32FC2);
    dft(merge_output_host, dft_output_host);

    dft_output_cuda.upload(dft_output_host);
//
    startsplit1 = std::chrono::system_clock::now();
    splitfilter<<<dimgrid3,dimblock3>>>(dft_output_cuda, split_output_cuda);
    hipDeviceSynchronize();
    endsplit1 = std::chrono::system_clock::now();
    elapsedtime += endsplit1 - startsplit1;
//
//
    startpadf = std::chrono::system_clock::now();
    pow_add_div_filter<<<dimgrid3,dimblock3>>>(split_output_cuda, pow_add_div_output_cuda, 1.0 / double(snr));
    hipDeviceSynchronize();
    endpadf = std::chrono::system_clock::now();
    elapsedtime += endpadf - startpadf;
//

    Mat pow_add_div_output_host(h.rows,h.cols,CV_32FC1);
    pow_add_div_output_cuda.download(pow_add_div_output_host);

    h_flat.release(); fftshift_output_host.release();
    /**************** End - Wiener Filter ********************************************/

    /**************** Start - Image Restoration using Filter *****************************************/
    cuda::GpuMat blurIn_cuda(roi.height, roi.width, CV_32FC1);
    cuda::GpuMat complexI(roi.height, roi.width, CV_32FC2);
    cuda::GpuMat complexH(roi.height, roi.width, CV_32FC2);
    cuda::GpuMat complexIH_split_cuda(roi.height, roi.width, CV_32FC1);
    cuda::GpuMat imgOut_cuda(roi.height, roi.width, CV_32FC1);
    cuda::GpuMat imgOut_norm_cuda(roi.height, roi.width, CV_8U);

    Mat complexI_host(roi.height, roi.width,CV_32FC2);
    Mat complexH_host(roi.height, roi.width,CV_32FC2);
    Mat complexIH_split_host(roi.height, roi.width,CV_32FC1);
    Mat imgOut_norm_host(roi.height, roi.width,CV_8U);

    blurIn_cuda.upload(blurIn);
//
    startmerge2 = std::chrono::system_clock::now();
    mergefilter<uint8_t, float2><<<dimgrid3,dimblock3>>>(blurIn_cuda, complexI);
    hipDeviceSynchronize();
    endmerge2 = std::chrono::system_clock::now();
    elapsedtime += endmerge2 - startmerge2;
//
    complexI.download(complexI_host);
    startdft = std::chrono::system_clock::now();
    dft(complexI_host, complexI_host, DFT_SCALE);
    enddft = std::chrono::system_clock::now();
    elapsedtime += enddft - startdft;
    complexI.upload(complexI_host);
//
    startmerge3 = std::chrono::system_clock::now();
    mergefilter<float, float2><<<dimgrid3,dimblock3>>>(pow_add_div_output_cuda, complexH);
    hipDeviceSynchronize();
    endmerge3 = std::chrono::system_clock::now();
    elapsedtime += endmerge3 - startmerge3;
//
    complexH.download(complexH_host);

    cuda::GpuMat complexIH(roi.height, roi.width, CV_32FC2);
    Mat complexIH_host(roi.height, roi.width,CV_32FC2);
//
    startmulspec = std::chrono::system_clock::now();
    mulSpectrums<<<dimgrid3,dimblock3>>>(complexI, complexH, complexIH);
    hipDeviceSynchronize();
    endmulspec = std::chrono::system_clock::now();
    elapsedtime += endmulspec - startmulspec;
//
    complexIH.download(complexIH_host);
    startidft = std::chrono::system_clock::now();
    idft(complexIH_host, complexIH_host);
    endidft = std::chrono::system_clock::now();
    elapsedtime += endidft - startidft;
    complexIH.upload(complexIH_host);
//
    startsplit2 = std::chrono::system_clock::now();
    splitfilter<<<dimgrid3,dimblock3>>>(complexIH, complexIH_split_cuda);
    hipDeviceSynchronize();
    endsplit2 = std::chrono::system_clock::now();
    elapsedtime += endsplit2 - startsplit2;
//
    complexIH_split_cuda.download(complexIH_split_host);
    /**************** End - Image Restoration using Filter ********************************************/


    /**************** Start - Normalization ********************************************/
    complexIH_split_host.convertTo(imgOut, CV_8U);
    imgOut_cuda.upload(imgOut);
    double min, max;
    minMaxLoc(imgOut, &min, &max);
//
    startimgnorm = std::chrono::system_clock::now();
    normalize_img<<<dimgrid3,dimblock3>>>(imgOut_cuda, imgOut_norm_cuda, min, max);
    hipDeviceSynchronize();
    endimgnorm = std::chrono::system_clock::now();
    elapsedtime += endimgnorm - startimgnorm;
//
    imgOut_norm_cuda.download(imgOut_norm_host);
    imwrite("final_images/restored_cuda_256.jpg", imgOut_norm_host);
    /**************** End - Normalization ********************************************/

    printf("\nExecution time: %f seconds\n",elapsedtime.count());

    /**************** Metrics - Image restoration ************************************/
    double psnr1, mse1;
    double psnr2, mse2;
    getPSNR(imgOut_norm_host, imgIn, R, snr, &mse1, &psnr1);
    getPSNR(imgIn, blurIn, R, snr, &mse2, &psnr2);
    printf("\nPerformance Metrics - Image Restoration:");
    printf("\n-----------------------------------------");
    printf("\nBetween Input image and Restored Image");
    printf("\nMean Squared Error: %f | Peak Signal to Noise Ratio: %f",mse1,psnr1);
    printf("\nBetween Input image and Blurred Image");
    printf("\nMean Squared Error: %f | Peak Signal to Noise Ratio: %f\n\n",mse2,psnr2);
}
